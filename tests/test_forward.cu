#include <cmath>
#include <fmha_api.h>
//#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

void test_fwd_with_mask() {
    int batch_size = 1;
    int nheads = 1;
    int headdim = 16;
    int max_seqlen_q_ = 128; 
    int max_seqlen_k_ = 128;
    
    float softmax_scale = 0.1;
    
    bool zero_tensors = false;
    bool is_causal = false;
    bool return_softmax = false;

    // q -> [bs * seq, head, head_dim]
    // q -> [1 * 128, 1, 16]
    // block q -> [128, 16]

    // k -> [bs * seq, head, head_dim]
    // k -> [1 * 128, 1, 16]
    // block k -> [128, 16]

    // v -> [bs * seq, head, head_dim]
    // v -> [1 * 128, 1, 16]
    // block k -> [128, 16]
    
    at::Tensor q_cpu = at::zeros({batch_size * max_seqlen_k_ * max_seqlen_k_, nheads, headdim}, at::kHalf);
    at::Tensor k_cpu = at::zeros({batch_size * max_seqlen_k_ * max_seqlen_k_, nheads, headdim}, at::kHalf);
    at::Tensor v_cpu = at::zeros({batch_size * max_seqlen_k_ * max_seqlen_k_, nheads, headdim}, at::kHalf);
  
    int cnt = 0;
    for (int i = 0; i < batch_size * max_seqlen_k_ * max_seqlen_k_; i ++) {
    	for (int j = 0; j < nheads; j ++) {
            for (int k = 0; k < headdim; k ++) {
                q_cpu[i][j][k] = cnt * 0.001;
                k_cpu[i][j][k] = cnt * 0.001;
                v_cpu[i][j][k] = cnt * 0.001;
                cnt ++;
            }
	}
    }

    auto q = q_cpu.cuda();
    auto k = k_cpu.cuda();
    auto v = v_cpu.cuda();

    at::Tensor cu_seqlens_q_cpu = at::zeros({batch_size * max_seqlen_k_ + 1}, at::kInt);
    at::Tensor cu_seqlens_k_cpu = at::zeros({batch_size * max_seqlen_k_ + 1}, at::kInt);
    
    for (int i = 0; i < batch_size * max_seqlen_k_ + 1; ++i) {
        cu_seqlens_q_cpu[i] = i * max_seqlen_q_;
        cu_seqlens_k_cpu[i] = i * max_seqlen_k_;
    }
    
    auto cu_seqlens_q = cu_seqlens_q_cpu.cuda();
    auto cu_seqlens_k = cu_seqlens_k_cpu.cuda();
    
    at::Tensor attn_mask = at::ones({batch_size * max_seqlen_k_, nheads, max_seqlen_q_, max_seqlen_k_}, at::kHalf).cuda();

    cnt = 0;
    for (int i = 0; i < batch_size * max_seqlen_k_; i ++) {
    	for (int j = 0; j < nheads; j ++) {
            for (int k = 0; k < max_seqlen_q_; k ++) {
                for (int l = 0; l < max_seqlen_k_; l ++) {
                    attn_mask[i][j][k][l] = cnt * 0.001;
                    cnt ++;
                }
            }
	    }
    }
    
    c10::optional<at::Generator> gen_;
    c10::optional<at::Tensor> attn_bias;

    // std::cout << "attn bias" << attn_bias << std::endl;

    std::vector<at::Tensor> ret = mha_fwd(
            q,         // total_q x num_heads x head_size, total_q := \sum_{i=0}^{b} s_i
            k,         // total_k x num_heads x head_size, total_k := \sum_{i=0}^{b} s_i
            v,         // total_k x num_heads x head_size, total_k := \sum_{i=0}^{b} s_i
            cu_seqlens_q,  // b + 1
            cu_seqlens_k,  // b + 1
            max_seqlen_q_,
            max_seqlen_k_,
            0.0,
            softmax_scale,
            zero_tensors,
            is_causal,
            return_softmax,
            gen_,
            attn_mask,
            attn_bias
	    );

    std::cout << "Ret vec size is " << ret.size();
    for (int i = 0; i < ret.size(); i ++) {
        ret[i].cpu();
        std::cout << ret[i] << std::endl;
    }
}


void test_fwd_with_mask_mini() {
    int batch_size = 1;
    int nheads = 1;
    int headdim = 16;
    int max_seqlen_q_ = 2; 
    int max_seqlen_k_ = 2;
    
    float softmax_scale = 0.1;
    
    bool zero_tensors = false;
    bool is_causal = false;
    bool return_softmax = false;

    // q -> [bs * seq, head, head_dim]
    // q -> [1 * 128, 1, 16]
    // block q -> [128, 16]

    // k -> [bs * seq, head, head_dim]
    // k -> [1 * 128, 1, 16]
    // block k -> [128, 16]

    // v -> [bs * seq, head, head_dim]
    // v -> [1 * 128, 1, 16]
    // block k -> [128, 16]
    
    at::Tensor q_cpu = at::zeros({batch_size * max_seqlen_k_ * max_seqlen_k_, nheads, headdim}, at::kHalf);
    at::Tensor k_cpu = at::zeros({batch_size * max_seqlen_k_ * max_seqlen_k_, nheads, headdim}, at::kHalf);
    at::Tensor v_cpu = at::zeros({batch_size * max_seqlen_k_ * max_seqlen_k_, nheads, headdim}, at::kHalf);
  
    int cnt = 0;
    for (int i = 0; i < batch_size * max_seqlen_k_ * max_seqlen_k_; i ++) {
    	for (int j = 0; j < nheads; j ++) {
            for (int k = 0; k < headdim; k ++) {
                q_cpu[i][j][k] = cnt * 0.001;
                k_cpu[i][j][k] = cnt * 0.001;
                v_cpu[i][j][k] = cnt * 0.001;
                cnt ++;
            }
	    }
    }

    auto q = q_cpu.cuda();
    auto k = k_cpu.cuda();
    auto v = v_cpu.cuda();

    at::Tensor cu_seqlens_q_cpu = at::zeros({batch_size * max_seqlen_k_ + 1}, at::kInt);
    at::Tensor cu_seqlens_k_cpu = at::zeros({batch_size * max_seqlen_k_ + 1}, at::kInt);
    
    for (int i = 0; i < batch_size * max_seqlen_k_ + 1; ++i) {
        cu_seqlens_q_cpu[i] = i * max_seqlen_q_;
        cu_seqlens_k_cpu[i] = i * max_seqlen_k_;
    }
    
    auto cu_seqlens_q = cu_seqlens_q_cpu.cuda();
    auto cu_seqlens_k = cu_seqlens_k_cpu.cuda();
    
    at::Tensor attn_mask_cpu = at::zeros({batch_size * max_seqlen_k_, nheads, max_seqlen_q_, max_seqlen_k_}, at::kHalf);

    cnt = 0;
    for (int i = 0; i < batch_size * max_seqlen_k_; i ++) {
    	for (int j = 0; j < nheads; j ++) {
            for (int k = 0; k < max_seqlen_q_; k ++) {
                for (int l = 0; l < max_seqlen_k_; l ++) {  
                    // if (l == 0) attn_mask[i][j][k][l] = -INFINITY;
                    if (l == 0) attn_mask_cpu[i][j][k][l] = -3e4;
                    else attn_mask_cpu[i][j][k][l] = 0;

                    attn_mask_cpu[i][j][k][l] = -3e4;
                    printf("i=%d, j=%d, k=%d, l=%d attn_mask=%f\n", i, j, k, l, attn_mask_cpu[i][j][k][l]);
                }
            }
	    }
    }

    auto attn_mask = attn_mask_cpu.cuda();

    c10::optional<at::Generator> gen_;
    c10::optional<at::Tensor> attn_bias;

    // std::cout << "attn bias: " << attn_bias << std::endl; 

    std::vector<at::Tensor> ret = mha_fwd(
            q,         // total_q x num_heads x head_size, total_q := \sum_{i=0}^{b} s_i
            k,         // total_k x num_heads x head_size, total_k := \sum_{i=0}^{b} s_i
            v,         // total_k x num_heads x head_size, total_k := \sum_{i=0}^{b} s_i
            cu_seqlens_q,  // b + 1
            cu_seqlens_k,  // b + 1
            max_seqlen_q_,
            max_seqlen_k_,
            0.0,
            softmax_scale,
            zero_tensors,
            is_causal,
            return_softmax,
            gen_,
            attn_mask,
            attn_bias
	    );

    // ret: std::vector<at::Tensor> result = {o, softmax_lse};
    // [bs * seq * seq, head, head_dim]
    // [1 * 2 * 2, 1, 16]
    std::cout << "Ret vec size is " << ret.size();
    for (int i = 0; i < ret.size(); i ++) {
        ret[i].cpu();
        std::cout << ret[i] << std::endl;
    }
}


void test_fwd_with_bias_mini() {
    int batch_size = 1;
    int nheads = 1;
    int headdim = 16;
    int max_seqlen_q_ = 2; 
    int max_seqlen_k_ = 2;
    
    float softmax_scale = 0.1;
    
    bool zero_tensors = false;
    bool is_causal = false;
    bool return_softmax = false;

    // q -> [bs * seq, head, head_dim]
    // q -> [1 * 128, 1, 16]
    // block q -> [128, 16]

    // k -> [bs * seq, head, head_dim]
    // k -> [1 * 128, 1, 16]
    // block k -> [128, 16]

    // v -> [bs * seq, head, head_dim]
    // v -> [1 * 128, 1, 16]
    // block k -> [128, 16]
    
    at::Tensor q_cpu = at::zeros({batch_size * max_seqlen_k_ * max_seqlen_k_, nheads, headdim}, at::kHalf);
    at::Tensor k_cpu = at::zeros({batch_size * max_seqlen_k_ * max_seqlen_k_, nheads, headdim}, at::kHalf);
    at::Tensor v_cpu = at::zeros({batch_size * max_seqlen_k_ * max_seqlen_k_, nheads, headdim}, at::kHalf);
  
    int cnt = 0;
    for (int i = 0; i < batch_size * max_seqlen_k_ * max_seqlen_k_; i ++) {
    	for (int j = 0; j < nheads; j ++) {
            for (int k = 0; k < headdim; k ++) {
                q_cpu[i][j][k] = cnt * 0.001;
                k_cpu[i][j][k] = cnt * 0.001;
                v_cpu[i][j][k] = cnt * 0.001;
                cnt ++;
            }
	    }
    }

    auto q = q_cpu.cuda();
    auto k = k_cpu.cuda();
    auto v = v_cpu.cuda();

    at::Tensor cu_seqlens_q_cpu = at::zeros({batch_size * max_seqlen_k_ + 1}, at::kInt);
    at::Tensor cu_seqlens_k_cpu = at::zeros({batch_size * max_seqlen_k_ + 1}, at::kInt);
    
    for (int i = 0; i < batch_size * max_seqlen_k_ + 1; ++i) {
        cu_seqlens_q_cpu[i] = i * max_seqlen_q_;
        cu_seqlens_k_cpu[i] = i * max_seqlen_k_;
    }
    
    auto cu_seqlens_q = cu_seqlens_q_cpu.cuda();
    auto cu_seqlens_k = cu_seqlens_k_cpu.cuda();
    
    at::Tensor attn_bias_cpu = at::zeros({batch_size * max_seqlen_k_, nheads, max_seqlen_q_, max_seqlen_k_}, at::kHalf);

    cnt = 0;
    for (int i = 0; i < batch_size * max_seqlen_k_; i ++) {
    	for (int j = 0; j < nheads; j ++) {
            for (int k = 0; k < max_seqlen_q_; k ++) {
                for (int l = 0; l < max_seqlen_k_; l ++) {  
                    // if (l == 0) attn_mask[i][j][k][l] = -INFINITY;
                    if (l == 0) attn_bias_cpu[i][j][k][l] = -3e4;
                    else attn_bias_cpu[i][j][k][l] = 0;

                    attn_bias_cpu[i][j][k][l] = 100;
                    printf("i=%d, j=%d, k=%d, l=%d attn_bias=%f\n", i, j, k, l, attn_bias_cpu[i][j][k][l]);
                    // std::cout << "i=" << i << ", j=" << j << ", k=" << k << ", l" 
                    //     << l << << ", attn_bias=" << attn_bias_cpu[i][j][k][l] << std::endl;
                }
            }
	    }
    }

    auto attn_bias = attn_bias_cpu.cuda();

    c10::optional<at::Generator> gen_;
    c10::optional<at::Tensor> attn_mask;

    // std::cout << attn_mask << std::endl;

    std::vector<at::Tensor> ret = mha_fwd(
            q,         // total_q x num_heads x head_size, total_q := \sum_{i=0}^{b} s_i
            k,         // total_k x num_heads x head_size, total_k := \sum_{i=0}^{b} s_i
            v,         // total_k x num_heads x head_size, total_k := \sum_{i=0}^{b} s_i
            cu_seqlens_q,  // b + 1
            cu_seqlens_k,  // b + 1
            max_seqlen_q_,
            max_seqlen_k_,
            0.0,
            softmax_scale,
            zero_tensors,
            is_causal,
            return_softmax,
            gen_,
            attn_mask,
            attn_bias
	    );

    // ret: std::vector<at::Tensor> result = {o, softmax_lse};
    // [bs * seq * seq, head, head_dim]
    // [1 * 2 * 2, 1, 16]
    std::cout << "Ret vec size is " << ret.size();
    for (int i = 0; i < ret.size(); i ++) {
        ret[i].cpu();
        std::cout << ret[i] << std::endl;
    }
}


void test_fwd_with_bias() {
    int batch_size = 1;
    int nheads = 1;
    int headdim = 16;
    int max_seqlen_q_ = 128; 
    int max_seqlen_k_ = 128;
    
    float softmax_scale = 0.1;
    
    bool zero_tensors = false;
    bool is_causal = false;
    bool return_softmax = false;

    // q -> [bs * seq, head, head_dim]
    // q -> [1 * 128, 1, 16]
    // block q -> [128, 16]

    // k -> [bs * seq, head, head_dim]
    // k -> [1 * 128, 1, 16]
    // block k -> [128, 16]

    // v -> [bs * seq, head, head_dim]
    // v -> [1 * 128, 1, 16]
    // block k -> [128, 16]
    
    at::Tensor q_cpu = at::zeros({batch_size * max_seqlen_k_ * max_seqlen_k_, nheads, headdim}, at::kHalf);
    at::Tensor k_cpu = at::zeros({batch_size * max_seqlen_k_ * max_seqlen_k_, nheads, headdim}, at::kHalf);
    at::Tensor v_cpu = at::zeros({batch_size * max_seqlen_k_ * max_seqlen_k_, nheads, headdim}, at::kHalf);
  
    int cnt = 0;
    for (int i = 0; i < batch_size * max_seqlen_k_ * max_seqlen_k_; i ++) {
    	for (int j = 0; j < nheads; j ++) {
            for (int k = 0; k < headdim; k ++) {
                q_cpu[i][j][k] = cnt * 0.001;
                k_cpu[i][j][k] = cnt * 0.001;
                v_cpu[i][j][k] = cnt * 0.001;
                cnt ++;
            }
	    }
    }

    auto q = q_cpu.cuda();
    auto k = k_cpu.cuda();
    auto v = v_cpu.cuda();

    at::Tensor cu_seqlens_q_cpu = at::zeros({batch_size * max_seqlen_k_ + 1}, at::kInt);
    at::Tensor cu_seqlens_k_cpu = at::zeros({batch_size * max_seqlen_k_ + 1}, at::kInt);
    
    for (int i = 0; i < batch_size * max_seqlen_k_ + 1; ++i) {
        cu_seqlens_q_cpu[i] = i * max_seqlen_q_;
        cu_seqlens_k_cpu[i] = i * max_seqlen_k_;
    }
    
    auto cu_seqlens_q = cu_seqlens_q_cpu.cuda();
    auto cu_seqlens_k = cu_seqlens_k_cpu.cuda();
    
    at::Tensor attn_bias_cpu = at::zeros({batch_size * max_seqlen_k_, nheads, max_seqlen_q_, max_seqlen_k_}, at::kHalf);

    cnt = 0;
    for (int i = 0; i < batch_size * max_seqlen_k_; i ++) {
    	for (int j = 0; j < nheads; j ++) {
            for (int k = 0; k < max_seqlen_q_; k ++) {
                for (int l = 0; l < max_seqlen_k_; l ++) {  
                    // if (l == 0) attn_mask[i][j][k][l] = -INFINITY;
                    // if (l == 0) attn_bias_cpu[i][j][k][l] = -3e4;
                    // else attn_bias_cpu[i][j][k][l] = 0;
                    
                    attn_bias_cpu[i][j][k][l] = 0;
                    // attn_bias_cpu[i][j][k][l] = cnt * 0.001;
                    cnt ++;
                    // printf("i=%d, j=%d, k=%d, l=%d attn_bias=%f\n", i, j, k, l, attn_bias_cpu[i][j][k][l]);
                    // std::cout << "i=" << i << ", j=" << j << ", k=" << k << ", l" 
                    //     << l << << ", attn_bias=" << attn_bias_cpu[i][j][k][l] << std::endl;
                }
            }
	    }
    }

    auto attn_bias = attn_bias_cpu.cuda();

    c10::optional<at::Generator> gen_;
    c10::optional<at::Tensor> attn_mask;

    // std::cout << attn_mask << std::endl;

    std::vector<at::Tensor> ret = mha_fwd(
            q,         // total_q x num_heads x head_size, total_q := \sum_{i=0}^{b} s_i
            k,         // total_k x num_heads x head_size, total_k := \sum_{i=0}^{b} s_i
            v,         // total_k x num_heads x head_size, total_k := \sum_{i=0}^{b} s_i
            cu_seqlens_q,  // b + 1
            cu_seqlens_k,  // b + 1
            max_seqlen_q_,
            max_seqlen_k_,
            0.0,
            softmax_scale,
            zero_tensors,
            is_causal,
            return_softmax,
            gen_,
            attn_mask,
            attn_bias
	    );

    // ret: std::vector<at::Tensor> result = {o, softmax_lse};
    // [bs * seq * seq, head, head_dim]
    // [1 * 2 * 2, 1, 16]
    std::cout << "Ret vec size is " << ret.size();
    for (int i = 0; i < ret.size(); i ++) {
        ret[i].cpu();
        std::cout << ret[i] << std::endl;
    }
}

int main(){
    // test_fwd();
    // test_fwd_with_bias_mini();
    test_fwd_with_bias();
    return 0;
}
